#include "hip/hip_runtime.h"
#include<iostream>
#include<stdio.h>

__global__ void GPUFunction() {
   // std::cout << "Thread: " << threadIdx.x  << " running in block: " << blockIdx.x << std::endl;
   printf("Thread:%d running in block: %d\n", threadIdx.x, blockIdx.x);
}

void CPUFunction() {
   std::cout << "CPU Execution\n";
}

hipDeviceProp_t getGPUDeviceSettings() {
   int deviceID;
   hipGetDevice(&deviceID);
   hipDeviceProp_t props;
   hipGetDeviceProperties(&props, deviceID);

   std::cout << props.name << ": " << deviceID << std::endl;

   return props;
}

hipDeviceProp_t getCPUDeviceSettings() {
   hipDeviceProp_t props;
   hipGetDeviceProperties(&props, hipCpuDeviceId);

   std::cout << props.name << ": " << hipCpuDeviceId << std::endl;
   return props;
}


void printDeviceProps(hipDeviceProp_t& props) {
   std::cout << props.name << '\n';
   std::cout << "Settings: \n";

   std::cout << "Multitprocessor Count: " << props.multiProcessorCount << '\n'; 
   std::cout << "Warp size: " << props.warpSize << '\n';
   std::cout << "Major Compute Capability: " << props.major << '\n';
   std::cout << "Minor Compute Capability: " << props.minor << '\n';
   std::cout << "Clock Rate: " << props.clockRate << '\n';

   std::cout << std::endl; // clean the buffer.
}

int main() {
   hipDeviceProp_t gpu_props = getGPUDeviceSettings();
   hipDeviceProp_t cpu_props = getCPUDeviceSettings();

   printDeviceProps(gpu_props);
   printDeviceProps(cpu_props);

   CPUFunction();
   GPUFunction<<<2,gpu_props.warpSize>>>();
   std::cout << "This line is written in the vim editor \n";
   hipDeviceSynchronize();
}
