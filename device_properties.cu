#include<hip/hip_runtime.h>
#include<>

#include"utils/cuda_utils.cuh"
#include<stdio.h>


int main() {
	
	// Getting the number of supported devices
	int deviceCount = 0;
	cuda_assert(hipGetDeviceCount(&deviceCount));

	if (!deviceCount) {
		printf("No CUDA supported device found\n");
		return 0;
	}

	printf("CUDA Supported devices: %d", deviceCount);

	// getting active device 
	int activeDevice = 0;
	cuda_assert(hipGetDevice(&activeDevice));

	// getting device properties
	hipDeviceProp_t props;
	cuda_assert(hipGetDeviceProperties(&props, activeDevice));

	printDeviceInfo(props);

	return 0;
}