#include "hip/hip_runtime.h"
#include <stdio.h>

#include "utils/cuda_utils.cuh"
#include<stdio.h>

__global__ void readFromArray(const int* arr, int size) {
	int index = threadIdx.x + (blockDim.x * blockIdx.x);

	if (index >= size) return;

	printf("%d\n", arr[index]);
}

int main() {
    // Allocating host buffer
    int n = 40;
    int *array = (int*)malloc(sizeof(*array) * n);

    // Initialising the host array
    for(int i = 0; i < n; i++) array[i] = i+1;

    // Allocating device buffer
    int *d_array;
    cuda_assert(hipMalloc((void**)&d_array, sizeof(*array) * n));

    // Host to device data transfer
    cuda_assert(hipMemcpy((void*)d_array, (void*)array, sizeof(*array) * n, hipMemcpyHostToDevice));

    // Getting device props to determine warp size
    hipDeviceProp_t props = getGPUDeviceSettings();

    unsigned int no_of_blocks = ceil((float)n / props.warpSize);
    printf("No of blocks: %u\n", no_of_blocks);
    readFromArray<<<no_of_blocks, props.warpSize>>>(d_array, n);

    cuda_assert(hipDeviceSynchronize());
    return 0;

	// We should create threads in the multiples of 32 for boosted performance on the cuda SM's
	// But our array size might not be in the multiple of 32, and sometimes may be more than a multiple
	
	// To benefit from the performance boost of thread count in multiple of 32, we should ensure that the 
	// extra threads do not access the overflowed memory.
	// To prevent this, we can pass in the size of the array into the function, and we can put a check
	// in the function to see if the index is in valid bounds.
}
